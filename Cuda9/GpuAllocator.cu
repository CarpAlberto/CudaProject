#include "GpuAllocator.h"

using namespace gpuNN;


GpuAllocator::GpuAllocator(const std::size_t totalSize):
	BaseAllocator(totalSize)
{
}

void* GpuAllocator::Allocate(const std::size_t size, const std::size_t alignment) {

	void* devPtr = 0;
	hipError_t error = hipMalloc(&devPtr, size);
	if (error != hipError_t::hipSuccess) {
		throw new MemoryAllocationException("Cuda Failed to allocate memory");
	}
	else 
	{
		m_offset += size;
		this->points[devPtr] = (double)size;
		return devPtr;
	}
}

void GpuAllocator::Free(void* ptr)
{
	hipFree(&ptr);
}
void GpuAllocator::Reset()
{
	m_offset = 0;

}
void GpuAllocator::Init()
{

}

GpuAllocator::~GpuAllocator()
{
	Reset();
}
